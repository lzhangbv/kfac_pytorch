#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <vector>

#include "tcmm_kernel.h"

using namespace std;

static hipsolverHandle_t g_cusolverH = NULL;
static hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;


#define checkKernelErrors(expr) do {                                                        \
    expr;                                                                                   \
                                                                                            \
    hipError_t __err = hipGetLastError();                                                 \
    if (__err != hipSuccess) {                                                             \
        printf("Line %d: '%s' failed: %s\n", __LINE__, # expr, hipGetErrorString(__err));  \
        abort();                                                                            \
    }                                                                                       \
} while(0)

hipsolverHandle_t get_cusolver_handler() {
    if (g_cusolverH == NULL) {
        cusolver_status = hipsolverDnCreate(&g_cusolverH);
        assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    }
    return g_cusolverH;
}


std::vector<torch::Tensor> tcmm_symeig(torch::Tensor a) {
    const auto a_shape = a.sizes();
    const int m = a_shape[0];
    const int lda = m;
    int lwork = 0;
    int *devInfo = NULL;
    hipError_t cudaStat1 = hipSuccess;

    auto options_float =
        torch::TensorOptions()
        .dtype(a.dtype())
        .layout(torch::kStrided)
        .device(a.device().type())
        .requires_grad(false);

    auto A = a.data_ptr<float>();
    auto V = torch::zeros({m, m}, options_float).copy_(a); // eigenvectors
    //auto V = a.copy_(a);
    auto W = torch::zeros({m}, options_float); // eigenvalues

    hipsolverHandle_t cusolverH = get_cusolver_handler();
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;

    cusolver_status = hipsolverDnSsyevd_bufferSize(
            cusolverH,
            jobz,
            uplo,
            m,
            V.data_ptr<float>(),
            lda,
            W.data_ptr<float>(),
            &lwork);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
    float *d_work = NULL;
    cudaStat1 = hipMalloc((void**)&d_work, sizeof(float)*lwork);
    assert(hipSuccess == cudaStat1);
    cudaStat1 = hipMalloc ((void**)&devInfo, sizeof(int));
    assert(hipSuccess == cudaStat1);

    cusolver_status = hipsolverDnSsyevd(
            cusolverH,
            jobz,
            uplo,
            m,
            V.data_ptr<float>(),
            lda,
            W.data_ptr<float>(),
            d_work,
            lwork,
            devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);
    hipFree(devInfo);
    hipFree(d_work);
    std::vector<torch::Tensor> tuple;
    tuple.push_back(W); 
    tuple.push_back(V); 
    return tuple;
}

std::vector<torch::Tensor> tcmm_symeig_sparse(torch::Tensor a) {
    std::vector<torch::Tensor> tuple;
    tuple.push_back(a); 
    tuple.push_back(a); 
    return tuple;
}
